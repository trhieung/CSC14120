#include "hip/hip_runtime.h"
#include "./final.cuh"

// Kernel function to perform im2col on GPU
__global__ void im2col_kernel(float* image, float* data_col,
                               int height_in, int width_in,
                               int height_kernel, int width_kernel,
                               int height_out, int width_out,
                               int channel_in, int stride) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;

    if (idx < hw_out * channel_in) {
        int c = idx / hw_out; // Channel index
        int i = idx % hw_out;  // Output index

        int step_h = i / width_out;
        int step_w = i % width_out;
        int start_idx = step_h * width_in * stride + step_w * stride;  // Left-top idx of window

        for (int j = 0; j < hw_kernel; j++) {
            int cur_col = start_idx % width_in + j % width_kernel;
            int cur_row = start_idx / width_in + j / width_kernel;

            if (cur_col >= 0 && cur_col < width_in && cur_row >= 0 && cur_row < height_in) {
                int pick_idx = cur_row * width_in + cur_col;
                data_col[idx] = image[c * hw_in + pick_idx];
            } else {
                data_col[idx] = 0;
            }

            idx += channel_in * hw_out;  // Move to the next channel
        }
    }
}

// Wrapper function to call im2col_kernel from CPU
void im2col_gpu(float* image, float* data_col,
                int height_in, int width_in,
                int height_kernel, int width_kernel,
                int height_out, int width_out,
                int channel_in, int stride) {
    int hw_out = height_out * width_out;
    int blockSize = 256;
    int num_blocks = (channel_in * hw_out + num_threads - 1) / num_threads;
    im2col_kernel<<<num_blocks, blockSize>>>(image, data_col,
                                               height_in, width_in,
                                               height_kernel, width_kernel,
                                               height_out, width_out,
                                               channel_in, stride);
}

void check() {
    // CPU
    // Matrix data_col;
    // im2col(bottom.col(i), data_col);
    // data_cols[i] = data_col;

    // // GPU
    // float* d_image;
    // float* d_data_col;
    // hipMalloc((void**)&d_image, sizeof(float) * height_in * width_in * channel_in);
    // hipMalloc((void**)&d_data_col, sizeof(float) * height_out * width_out * channel_in * height_kernel * width_kernel);
    // hipMemcpy(d_image, bottom.col(i).data(), sizeof(float) * height_in * width_in * channel_in, hipMemcpyHostToDevice);

    // im2col_gpu(d_image, d_data_col, height_in, width_in, height_kernel, width_kernel, height_out, width_out, channel_in, stride);

    // hipMemcpy(data_cols[i].data(), d_data_col, sizeof(float) * height_out * width_out * channel_in * height_kernel * width_kernel, hipMemcpyDeviceToHost);

    // // Check equality (You may need to implement a function to compare CPU and GPU results)
    // // ...
    
    // hipFree(d_image);
    // hipFree(d_data_col);
}
