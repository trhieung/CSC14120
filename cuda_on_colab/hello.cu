#include "hip/hip_runtime.h"
#include <iostream>
#include "src/optimizer/sgd.h"

__global__ void myKernel(void) {
	printf("hello from kernel\n");
}

int main(void) {
	myKernel <<<2, 2>>>();
	hipDeviceSynchronize()
	printf("Hello CUDA!\n");
	return 0;
}