#include "hip/hip_runtime.h"
#include "./cuda_file.cuh"
#include "./Check.cuh"
#include <cstdio>

__global__ void cudaKernel() {
    printf("Hello from CUDA!\n");
}

void runCudaFunction() {
    CHECK(cudaKernel<<<2, 2>>>());
    CHECK(hipDeviceSynchronize());
}
