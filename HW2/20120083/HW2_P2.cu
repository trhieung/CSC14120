
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}
#define TILE_WIDTH 32
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void matrix_multiplication_kernel1(float* A, float* B, float* C, int m, int n, int k)
{
	//TODO
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float val = 0.0;
        for (int i = 0; i < n; ++i) {
            val += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = val;
    }
}

__global__ void matrix_multiplication_kernel2(float* A, float* B, float* C, int m, int n, int k)
{
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];
	//TODO

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;  

    float val = 0.0f;

    for (int i = 0; i < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++i) {
        if (row < m && i * TILE_WIDTH + tx < n) {
            s_A[ty][tx] = A[row * n + i * TILE_WIDTH + tx];
        } else {
            s_A[ty][tx] = 0.0;
        }

        if (i * TILE_WIDTH + ty < n && col < k) {
            s_B[ty][tx] = B[(i * TILE_WIDTH + ty) * k + col];
        } else {
            s_B[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; ++j) {
            val += s_A[ty][j] * s_B[j][tx];
        }

        __syncthreads();
    }

    if (row < m && col < k) {
        C[row * k + col] = val;
    }  
}

void matrix_multiplication(float* A, float* B, float* C, int m, int n, int k,
    bool useDevice = false, dim3 blockSize = dim3(1),int kernelType=1)
{
    GpuTimer timer;
    timer.Start();
    if (useDevice == false)
    {
        // TODO
        for (int i = 0; i < m; ++i) {
            for (int j = 0; j < k; ++j) {
                float val = 0.0;
                for (int x = 0; x < n; ++x) {
                    val += A[i * n + x] * B[x * k + j];
                }
                C[i * k + j] = val;
            }
        }
    }
    else // Use device
    {
        // TODO: Allocate device memories
        float* d_A, * d_B, * d_C;
        size_t size_A = m * n * sizeof(float);
        size_t size_B = n * k * sizeof(float);
        size_t size_C = m * k * sizeof(float);

        CHECK(hipMalloc((void**)&d_A, size_A));
        CHECK(hipMalloc((void**)&d_B, size_B));
        CHECK(hipMalloc((void**)&d_C, size_C));

        // Copy data to device memories
        CHECK(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));
        
        dim3 gridSize((k - 1) / blockSize.x + 1, (m - 1) / blockSize.y + 1); // TODO: Compute gridSize
        
		if (kernelType == 1)
			matrix_multiplication_kernel1<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);
		else if (kernelType == 2)
			matrix_multiplication_kernel2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

		CHECK(hipGetLastError());
        // TODO: Copy result from device memory
        CHECK(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

        // TODO: Free device memories
        CHECK(hipFree(d_A));
        CHECK(hipFree(d_B));
        CHECK(hipFree(d_C));
        
		printf("Grid size: %d * %d, block size: %d * %d\n", 
			gridSize.x,gridSize.y, blockSize.x,blockSize.y);
    }
    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time (%s): %f ms\n",
        useDevice == true ? "use device" : "use host", time);
}

float checkCorrectness(float * a1, float* a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)	
		err += abs(a1[i] - a2[i]);
	err /= n;
	return err;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
int main(int argc, char** argv)
{
	printDeviceInfo();
	
	//Declare variables
    float* h_A; // The A matrix
    float* h_B; // The B matrix
    float* h_C; // The output C matrix
    float* correct_C; // The output C matrix

    int m;    // number of rows in the matrix A
    int n; // number of columns in the matrix A, number of rows in the matrix B
    int k; // number of columns in the matrix B

    m = (1 << 10);
    n = (1 << 9);
    k = (1 << 10);

    // Set up input data
    h_A = (float*)malloc(m * n * sizeof(float));
    h_B = (float*)malloc(n * k * sizeof(float));
    h_C = (float*)malloc(m * k * sizeof(float));
    correct_C = (float*)malloc(m * k * sizeof(float));

    for (int i = 0; i < m; i++)
        for (int j = 0;j < n;j++)
            h_A[i*n+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
 
    for (int i = 0; i < n; i++)
        for (int j = 0;j < k;j++)
            h_B[i*k+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);


    // Add vectors (on host)
    matrix_multiplication(h_A,h_B,correct_C,m,n,k);
	printf("\n");

	dim3 blockSize(32, 32); // Default
	if (argc == 3)
	{
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
	} 
    // Add in1 & in2 on device
	printf("Basic Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,1);
	float err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f\n\n", err);

	printf("Shared memory Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,2);
	err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f", err);	
	
    free(h_A);
    free(h_B);
    free(h_C);
    free(correct_C);

    return 0;
}
